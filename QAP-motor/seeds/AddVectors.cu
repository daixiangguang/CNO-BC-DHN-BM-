#include "hip/hip_runtime.h"
﻿#include "AddVectors.h"
#include<stdio.h>

__global__ void addVectorsMask(double *devPtrA, double *devPtrB, double *devPtrC, int size)
{
    int i = threadIdx.x ;//+ blockIdx.x * blockDim.x;
   // if(i!= size)
       // return;

    devPtrC[i] = devPtrA[i] + devPtrB[i];
__syncthreads();
}

void addVectors(double *A, double *B, double *C, int size)
{
    double *devPtrA,*devPtrB,*devPtrC;
    hipMalloc(&devPtrA,sizeof(double)* size);
    hipMalloc(&devPtrB,sizeof(double)* size);
    hipMalloc(&devPtrC,sizeof(double)* size);

    hipMemcpy(devPtrA,A, sizeof(double)* size, hipMemcpyHostToDevice);
    hipMemcpy(devPtrB,B, sizeof(double)* size, hipMemcpyHostToDevice);
    addVectorsMask<<<1,size>>>(devPtrA,devPtrB, devPtrC, size);

    hipMemcpy(C,devPtrC, sizeof(double)* size, hipMemcpyDeviceToHost);
	
	double *d=(double *)malloc(sizeof(double)* size);
	hipMemcpy(d,devPtrC, sizeof(double)* size, hipMemcpyDeviceToHost);
    for(int i=0;i<size;i++)
	{
		printf("A=%f\n",A[i]);
	}

	free(d);

    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);

}