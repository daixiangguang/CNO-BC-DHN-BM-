#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include ""
#include "E:\Matlab R2018a\extern\include\mex.h"


#include <stdio.h>


#define N   150 //��������
#define P   3  //��������
#define POP 1 //��Ⱥ���� 
#define SIZE N*P*POP 
#define ALPHA 5 //�ͷ�����1
#define BETA  5//�ͷ�����2

//�������BLOCKS��HREAD_NUM 
#define BLOCKS POP
#define THREAD_NUM P //THREAD_NUM С�ڵ���POP����POP�ܳ���THREAD_NUM,THREAD_NUM���1024��THREAD_NUM�����32,һ�����ó�P*POP,���������1024������BLOCKS
//xתxt��HREAD_NUM����ʱ��ҪBLOCKS


#define PSO_THREAD_NUM 1 //����߳�����������ܳ���POP�����ó�POP
#define PSO_BLOCKS SIZE/PSO_THREAD_NUM 

#define RD_THREAD_NUM 1
#define RD_BLOCKS P*POP/RD_THREAD_NUM
#define W 1
#define C1 2
#define C2 2
#define STOPNUM 500

#define data "iris.txt"
#define U N/P
#define ITER 10
#define MAXITER 20000
//cublas����˷��ھ���,A[m,k], B[k,n],C[n,m],alpha=1.0,beta=0.0
//C=A*B
//hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);

//��ע
//���ڴ�Ƕȣ�c���Եľ����ǰ������У�cublas�ľ����ǰ�������
//x[n,p*pop],����n������������p�����pop����Ⱥ��С
//�����к�s1,�轫xת��xt[n*pop,p],I1[p,1],��������s1=xt*I1,s1[n*pop,1]
//�����к�s2,I2[1,n],s2=I2*x,s2[1,p*pop]
//����t=d*x,d[n,n],x[n,p*pop],t[p*pop,n]
//index[n,p*pop],�������¾���
void generate_rd_num(int* a)
{
	int i, j;

	int b[N];
	// int* b = (int*)malloc(sizeof(int) * N);
	for (i = 0; i < N; i++)
		a[i] = i;
	srand(time(NULL));
	int temp;
	for (i = 0; i < N; i++)
	{
		//j = (int)((double)((N - i) * rand()) / (RAND_MAX + 1.0));
		j = rand() % N;
		temp = a[i];
		a[i] = a[j];
		a[j] = temp;
	}
}

//�첽���£�����batch�����浽����index[n,p*pop]��index��ÿ���е���Ԫ����ͬ�����£�ÿ��ֻ���첽����
void generate_batch(int *index)
{
	int pos;
	int *block = (int*)malloc(sizeof(int) * N*P);
	int* rd = (int*)malloc(sizeof(int) * N);
	int i, j, loop;
	for (i = 0; i < N; i++)
		for (j = 0; j < P; j++)
			block[i*P + j] = (i*P + j * (P + 1)) % (N*P);

	//print_matrix_int(block, N, P);

	for (loop = 0; loop < POP; loop++)
	{
		generate_rd_num(rd);

		//for (int ii=0;ii<N;ii++)
		//	printf("%d ",rd[ii]);
		//	printf("\n");
		//	printf("\n");
		for (i = 0; i < N; i++)
			for (j = 0; j < P; j++)
			{
				index[i*P*POP + loop * P + j] = block[rd[i] * P + j];
			}
		//print_index(index, N, P,POP,loop);
	}
	free(rd);
	free(block);
}

__global__ static void tt()
{
}

//����ͬ������ 
__global__ static void parallel_updated_GPU(double *y,double *obj, double *lbest, double *lbestx, double*initialx, double *x, double *d, double *s1, double *s2, double *t, int *index, int gen, hiprandState *globalState)
{
	hiprandGenerator_t gen1;  //�������������
	int tid, i, k, b, j;
	int loop;
	//hiprandState state;
	//hiprand_init(seed, tid, 0, &state);
	tid = blockIdx.x *blockDim.x + threadIdx.x; //��ȡ�̺߳�0~blocks*THREAD_NUM-1
	int rd[N], temp, ped;
	hiprandState localState = globalState[tid];
	//tt<<<1,20>>>();
    int count=0;
	if (tid%P == 0)
	{

		//������һ�г�ʼ����
		int col = (hiprand(&localState)) % P;
		for (int i = 0; i < N; i++)
			rd[i] = i * P;
		//���ҵ�һ��

		for (int i = 0; i < N; i++)
		{


			j = (hiprand(&localState)) % N;
			temp = rd[i];
			rd[i] = rd[j];
			rd[j] = temp;
		}
		//printf("%d,%d\n", tid, rd[0]);
		//���������࣬��಻������P���Ҵ���P
		ped = P + 1;
		for (i = 0; i < N; i++)
			for (j = 0; j < P; j++)
				index[i*P*POP + tid + j] = (ped * j + rd[i]) % (N*P);
	}
	for (i = 0; i < N; i++)
	{
		x[i*P*POP + tid] = initialx[i*P*POP + tid];
	}
	__syncthreads();





	b = tid / P;
	double flag = 1.0;
	int it = 0;
	localState = globalState[tid / P];
	while (it < ITER)
	{
		flag = 0;
		//double total=0.0,const1=0.0,const2=0.0;
				//������һ�г�ʼ����
		for (int i = 0; i < N; i++)
			rd[i] = i;
		//���ҵ�һ��

		for (int i = 0; i < N; i++)
		{


			j = (hiprand(&localState)) % N;
			temp = rd[i];
			rd[i] = rd[j];
			rd[j] = temp;
		}
        double total = 0.0;
		for (loop = 0; loop < N; loop++)
		{

			temp = index[rd[loop] * POP*P + tid];
			i = temp / P;
			k = temp % P + b * P;

			//loopȡֵ0~N-1
			//int tid =  threadIdx.x;
			int pos = P * POP*i + k; //�����̺߳ż����index����
			double x_pos = x[pos];//��������ȡ��x�����ݣ�׼������x(pos)
			double half = 0.5;
			//double s1_i = s1[b*N + i];//����s1(i)
			double s1_i = s1[i*POP + b];//����s1(i)
			double s2_k = s2[k];//����s2(k)
			double dedx = 0.5*t[pos] + ALPHA * (s1_i - x_pos - 2 + P) + BETA * (s2_k - x_pos - 2 * U + N);
			//double dedx=t[k*N+i]+ALPHA*(s1_i-half-x_pos)+BETA*(s2_k-x_pos-U+half);
			//double dedx=ALPHA*(s1_i-half-x_pos)+BETA*(s2_k-x_pos-U+half);

			if (-dedx >= 0)
				x[pos] = 1.0;
			else
				x[pos] = -1.0;

			//��������Ż���if x[i][k]==0 t[k][j]=0
			//for(int j=0;j<N;j++)
				//t[k*N+j]=d[j*N+i]*x[pos];
			if (x[pos] != x_pos)
			{
				s1[i*POP + b] = s1_i - x_pos + x[pos];
				s2[k] = s2_k - x_pos + x[pos];
				double deltx = x[pos] - x_pos;
				for (int j = 0; j < N; j++)
					t[j*P*POP + k] = t[j*P*POP + k] + d[j*N + i] * deltx;
			}
			__syncthreads();
            if(tid==0)
            {
                double const1 = 0.0, const2 = 0.0;
                int ii,kk,id;
                for (kk = tid; kk < P + tid; kk++)
                for (ii = 0; ii < N; ii++)
                {
                    id = ii * P*POP + kk;
                        if (x[id] == 1.0)
                            total = total + t[id];
                        else
                            total = total - t[id];
                }
                for (int ii = 0; ii < N; ii++)
                    const1 = const1 + ALPHA * (s1[ii*POP + b] - 2 + P)*(s1[ii*POP + b] - 2 + P);
                for (int kk = 0; kk < P; kk++)
                    const2 = const2 + BETA * (s2[b*P + kk] - 2 * U + N)*(s2[b*P + kk] - 2 * U + N);
                total = 0.5*(total  + const1 + const2);
            }
			//flag = flag + (s1[i*POP + b] - 2 + P)*(s1[i*POP + b] - 2 + P) + (s2[k] - 2 * U + N)*(s2[k] - 2 * U + N);
		}
		__syncthreads();
        y[count]=total;
                count++;
		it++;
		//if (flag == 0)
		//	break;
	}
	//__syncthreads();
	if (tid%P == 0)
	{
		double total = 0.0, const1 = 0.0, const2 = 0.0;
		int id;
		for (k = tid; k < P + tid; k++)
			for (i = 0; i < N; i++)
			{
				id = i * P*POP + k;
				if (x[id] == 1.0)
					total = total + t[id];
				else
					total = total - t[id];
			}
		for (i = 0; i < N; i++)
			const1 = const1 + ALPHA * (s1[i*POP + b] - 2 + P)*(s1[i*POP + b] - 2 + P);
		for (k = 0; k < P; k++)
			const2 = const2 + BETA * (s2[b*P + k] - 2 * U + N)*(s2[b*P + k] - 2 * U + N);
		total = 0.5*(total  + const1 + const2);
		obj[tid / P] = total;
		//����ǵ�һ�ε�������ô������Ⱥ�õ���Ŀ�꺯��ֵ���϶������ŵ�
		if (gen == 0)
		{
			lbest[tid / P] = total;
			for (k = tid; k < tid + P; k++)
				for (i = 0; i < N; i++)
				{
					id = i * P*POP + k;
					lbestx[id] = x[id];
				}
		}
		else
		{
			if (obj[tid / P] < lbest[tid / P])
			{

				lbest[tid / P] = obj[tid / P];
				for (k = tid; k < tid + P; k++)
					for (i = 0; i < N; i++)
					{
						id = i * P*POP + k;
						lbestx[id] = x[id];
					}
			}
		}
	}



}


//���ļ���ȡd
void generate_d(double* d, char* str)
{
	FILE* fp;            /*�ļ�ָ��*/
	errno_t error;
	double* TemporaryD = d;
	error = fopen_s(&fp, str, "r");
	if (error != 0)
	{
		perror("fail to read");
		exit(1);
	}
	int i = 0;
	while (!feof(fp))
	{
		fscanf_s(fp, "%lf", &d[i++]);
		//fseek(fp, 1L, SEEK_CUR);   /*fpָ��ӵ�ǰλ������ƶ�*/
	}

	fclose(fp);                     //�ر��ļ�
}
void generate_s1(hipblasHandle_t handle, double* s1, double *x, double* I1)
{

	//���Դ��е�xת����xt,һ��THREAD_NUM����POP�����һ���Կ����߳���������POP������forѭ������
	//for (int loop = 0; loop < REPEAT; loop++)
	//	trans_x_xt << <1, TRANS_THREAD_NUM >> > (xt, x, loop);


	//x[n,p*sizepop]
	//����s1,s1=x*I1,x���кͱ��浽s1

	const double alpha = 1.0f;
	const double beta = 0.0f;
	//�����к�s1,s1=xt*I1,I1[p,1],�൱��xt[n*pop,p]��A[m,k]��I1[p,1]��B[k,n]��
	//hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, N*POP, P, &alpha, I1, 1, xt, P, &beta, s1, 1);
	//�����к�s1,s1=x*I1,I1[p,1],�൱��x[n,p*pop]��A[m,k]��I1[p*pop,pop]��B[k,n]��
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, POP, N, P*POP, &alpha, I1, POP, x, P*POP, &beta, s1, POP);
}

void generate_s2(hipblasHandle_t handle, double* s2, double* x, double* I2)
{
	//x[n,p*sizepop]
	//����s1,s1=x*I1,x���кͱ��浽s1


	const double alpha = 1.0f;
	const double beta = 0.0f;
	//����s2,s2=I2*x,x���кͱ��浽s2
	//�����к�s1,s2=I2*x,I2[1,n],�൱��I2[1,n]��A[m,k]��x[n,p*pop]��B[k,n]��
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, P*POP, 1, N, &alpha, x, P*POP, I2, N, &beta, s2, P*POP);

}

void generate_t(hipblasHandle_t handle, double* t, double* d, double* x)
{
	const double alpha = 1.0f;
	const double beta = 0.0f;
	//����t=d*x,d[n,n],x[n,p*pop],t[p*pop,n],�൱��d[n,n]��A[m,k]��x[n,p*pop]��B[k,n]��
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, P*POP, N, N, &alpha, x, P*POP, d, N, &beta, t, P*POP);

}

void generate_I1(double* I)
{
	int i, j;
	for (i = 0; i < POP; i++)
	{
		for (j = 0; j < P*POP; j++)
			I[j*POP + i] = 0.0;
		for (j = i * P; j < (i + 1)*P; j++)
			I[j*POP + i] = 1.0;
	}
}
void generate_I2(double* I, int num)
{
	int i;
	for (i = 0; i < num; i++)
		I[i] = 1.0;
}

void cp_initialx_x(double *x,double *initialx)
{
	int i;
	double rd;
	srand((unsigned)time(NULL));
	for (i = 0; i < SIZE; i++) {

		rd = 2.0 * rand() / RAND_MAX - 1; //����-1��1�������
		if (rd > 0)
			x[i] = 1.0;
		else
			x[i] = -1.0;
	}
}
void initial_x(double* x)
{
	int i;
	double rd;
	srand((unsigned)time(NULL));
	for (i = 0; i < SIZE; i++) {

		rd = 2.0 * rand() / RAND_MAX - 1; //����-1��1�������
		if (rd > 0)
			x[i] = 1.0;
		else
			x[i] = -1.0;
	}
}

void initial_obj(double* x)
{
	int i;
	for (i = 0; i < POP; i++)
		x[i] = 0.0;
}

//����ȫ������Ŀ�꺯��ֵ�����Ž�,
__global__ static void generate_global_best(double *y, double *gbest, double *gbestx, double * lbest, double *lbestx, int it)
{

	int i, j, k, flag = 0;
	//������Ⱥ0��Ŀ�꺯��ֵ��С�Ҿ������Ž�
	double gb = lbest[0];
	int id = 0;
	//��ȡȫ�����Ž�
	for (i = 0; i < POP; i++)
		if (lbest[i] < gb)
		{
			gb = lbest[i];
			id = i;
		}
	if (it == 0 || gb < *gbest)
	{
		*gbest = gb;
		y[it] = gb;
		k = 0;
		int count = 0;
		for (j = 0; j < N; j++)
			for (i = id * P; i < (id + 1)*P; i++)
				gbestx[k++] = lbestx[j*P*POP + i];

		/*
			for (j = 0; j < N; j++)
			{
				for (i = id * P; i < (id + 1)*P; i++)
					printf("%f ", lbestx[j*P*POP + i]);
				printf("\n");
			}*/
	}
	else
	{
		y[it] = *gbest;
	}

	//printf("%d=%lf\n", it, y[it]);


}
__global__ static void pso(double *x, double *initialx, double *initialv, double *gbestx, double *lbestx, double *rd1, double *rd2, double *rd3)
{

	int i, k;
	int tid = blockIdx.x *blockDim.x + threadIdx.x; //��ȡ�̺߳�0~blocks*THREAD_NUM-1


	//printf("%f %f\n", rd1,rd2);
	//double rd1 = 0.7;
	//double rd2 = 0.5;
	/*
	if (tid == 0)
	{
		printf("\n");
		for (i = 0; i < N; i++)
		{
			for (k = 0; k < P; k++)
				printf("% f", initialx[i*P + k]);
			printf("\n");
		}
	}*/
	i = tid / (P*POP);
	k = tid % (P*POP) % P;

	/*
	   initial_v{j}=w1*initial_v{j}+beta1*rd1*(pbest_x{j}-initial_x{j})+beta2*rd2*(zbest_x-initial_x{j});
	   initial_x{j}=initial_x{j}+initial_v{j};
	   initial_x{j}=round(min(1,max(0,initial_x{j})));  %for zero one
	*/


	/*
	initialv[tid] = W * initialv[tid] + C1 * rd1[tid % (P*POP) / P] * (lbestx[tid] - initialx[tid]) + C2 * rd2[tid % (P*POP) / P] * (gbestx[i*P + k] - initialx[tid]);
	initialx[tid] = initialx[tid] + initialv[tid];
	if (initialx[tid] <= -1.0)
		initialx[tid] = -1.0;
	if (initialx[tid] >= 1.0)
		initialx[tid] = 1.0;
	if (initialx[tid] >= 0)
		initialx[tid] = 1.0;
	else
		initialx[tid] = -1.0;*/


		//initialv[tid] = W * initialv[tid] + C1 * rd1[tid % (P*POP) / P] * ((lbestx[tid] + 1) / 2.0 - (initialx[tid] + 1) / 2.0) + C2 * rd2[tid % (P*POP) / P] * ((gbestx[i*P + k] + 1) / 2.0 - (initialx[tid] + 1) / 2.0);
	initialv[tid] = W * initialv[tid] + C1 * rd1[tid] * ((lbestx[tid] + 1) / 2.0 - (initialx[tid] + 1) / 2.0) + C2 * rd1[tid] * ((gbestx[i*P + k] + 1) / 2.0 - (initialx[tid] + 1) / 2.0);
	double s;


	s = 1 / (1 + exp(-initialv[tid]));

	if (s > rd3[tid])
		initialx[tid] = 1.0;
	else
		initialx[tid] = -1.0;

	//if (5 / (N*P) > rd2[tid])
		//initialx[tid] = 1 - 2 * (1 - (initialx[tid] + 1) / 2.0);


}

void initial_v(double* x)
{
	int i;
	double rd;
	srand((unsigned)time(NULL));
	for (i = 0; i < SIZE; i++) {

		rd = 2.0 * rand() / RAND_MAX - 1; //����-1��1�������
		x[i] = rd;
		/*
				if (rd > 0)
					x[i] =0;
				else
					x[i] = 0;
		*/
	}
}
__global__ void setup_kernel(hiprandState *state, unsigned long seed)
{
	int tid = blockIdx.x *blockDim.x + threadIdx.x; //��ȡ�̺߳�0~blocks*THREAD_NUM-1
	hiprand_init(seed, tid, 0, &state[tid]);// initialize the state
}
__global__ void generate_rd1_rd2(double *rd1, double *rd2, double *rd3, hiprandState *globalState)
{
	int tid = blockIdx.x *blockDim.x + threadIdx.x; //��ȡ�̺߳�0~blocks*THREAD_NUM-1
	int i;
	hiprandState localState;
	if (tid%P == 0)
	{
		//localState = globalState[tid / P];
		//rd1[tid / P] = hiprand_uniform(&localState);
		//rd2[tid / P] = hiprand_uniform(&localState);
	}
	localState = globalState[tid];
	for (i = 0; i < N; i++)
	{
		rd1[i*P*POP + tid] = hiprand_uniform(&localState);
		rd2[i*P*POP + tid] = hiprand_uniform(&localState);
		rd3[i*P*POP + tid] = hiprand_uniform(&localState);
		//printf("%d %lf", i*P*POP + tid, rd3[i*P*POP + tid]);
	}
}
__global__ void stop(double *y, int *is_stop, int it)
{
	*is_stop = 0;
	if (y[it] == y[it - STOPNUM])
		*is_stop = 1;
}
int Hopfield_syn_cuda(double *y,double *gbestx,double *initialx)
{
	
	//ѡ���Կ�,ubuntu�û���3��2080ti�Կ������ϴ�ѧ��2��rtx8000�Կ�
	hipSetDevice(1);


	double *xh, *xth, *xd, *xtd;
	xh = (double*)malloc(sizeof(double) * SIZE); //x���ڴ�����������ڴ�ռ�
	xth = (double*)malloc(sizeof(double)*SIZE); //xt���ڴ�����������ڴ�ռ�
	cp_initialx_x(xh,initialx);//printf("\n");print_matrix(xh, N, P*POP);//��ʼ��x
	hipMalloc((void**)&xd, sizeof(double) * SIZE); //x���Դ�����������Դ�ռ�
	hipMalloc((void**)&xtd, sizeof(double) * SIZE); //xtd���Դ�����������Դ�ռ�
	hipMemcpy(xd, xh, sizeof(int) * SIZE, hipMemcpyHostToDevice); //�ڴ��е�x���Ƶ��Դ�

	//���Կ����㣬��ʱ����#pragma omp parallel for num_threads(3)


	//����x��xt�Ƿ���ȷ����ʱ����
	/*print_matrix(xh, N, P*POP);
	hipMemcpy(xth, xtd, sizeof(int)*SIZE, hipMemcpyDeviceToHost); //�Դ����ݿ������ڴ�
	print_matrix(xth, N* POP, P);*/

	//����obj,index,I1��I2��s1��s2��t��d���ڴ����Դ���� 
	int *indexd, *indexh;
	double *sd1, *sd2, *sh1, *sh2, *Id1, *Id2, *Ih1, *Ih2, *td, *th, *dd, *dh;
	double *objd, *objh;
	double *initialxh, *initialxd, *initialvh, *initialvd;
	initialxh = (double*)malloc(sizeof(double)*SIZE);
	initialvh = (double*)malloc(sizeof(double)*SIZE); initial_v(initialvh);
	hipMalloc((void**)&initialvd, sizeof(double) * SIZE);
	indexh = (int*)malloc(sizeof(int) *SIZE); generate_batch(indexh);//index���ڴ�����������ڴ�ռ�
	Ih1 = (double*)malloc(sizeof(double)  * POP*P*POP); generate_I1(Ih1);//I1���ڴ�����������ڴ�ռ�
	Ih2 = (double*)malloc(sizeof(double) * N); generate_I2(Ih2, N);//I2���ڴ�����������ڴ�ռ�
	sh1 = (double*)malloc(sizeof(double) * N*POP);//s1���ڴ�����������ڴ�ռ�
	sh2 = (double*)malloc(sizeof(double) * P*POP);//s2���ڴ�����������ڴ�ռ�
	th = (double*)malloc(sizeof(double) * SIZE);//t���ڴ�����������ڴ�ռ�
	dh = (double*)malloc(sizeof(double) * N*N); generate_d(dh, data);//d���ڴ�����������ڴ�ռ�
	objh = (double*)malloc(sizeof(double) * POP); initial_obj(objh);//obj���ڴ�����������ڴ�ռ�

	hipMalloc((void**)&indexd, sizeof(int) * SIZE);//I1���ڴ�����������Դ�ռ�
	hipMalloc((void**)&Id1, sizeof(double) * POP*P*POP);//I1���ڴ�����������Դ�ռ�
	hipMalloc((void**)&Id2, sizeof(double) * N);//I2���Դ�����������Դ�ռ�
	hipMalloc((void**)&sd1, sizeof(double) * N*POP);//I1���ڴ�����������Դ�ռ�
	hipMalloc((void**)&sd2, sizeof(double) * P*POP);//I2���Դ�����������Դ�ռ�
	hipMalloc((void**)&td, sizeof(double) * SIZE);//t���Դ�����������Դ�ռ�
	hipMalloc((void**)&dd, sizeof(double) * N*N);//d���Դ�����������Դ�ռ�
	hipMalloc((void**)&objd, sizeof(double) * POP);//obj���Դ�����������Դ�ռ�
	hipMalloc((void**)&initialxd, sizeof(double) * SIZE);
	hipMemcpy(initialxd, xh, sizeof(double) * SIZE, hipMemcpyHostToDevice); //�ڴ��е�x���Ƶ��Դ�
	hipMemcpy(initialvd, initialvh, sizeof(double) * SIZE, hipMemcpyHostToDevice); //�ڴ��е�x���Ƶ��Դ�

	hipMemcpy(indexd, indexh, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(Id1, Ih1, sizeof(double) * POP*P*POP, hipMemcpyHostToDevice);
	hipMemcpy(Id2, Ih2, sizeof(double) * N, hipMemcpyHostToDevice);
	hipMemcpy(td, th, sizeof(double) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dd, dh, sizeof(double) * N*N, hipMemcpyHostToDevice);
	hipMemcpy(objd, objh, sizeof(double) * POP, hipMemcpyHostToDevice);



	double *yh, *yd, *gbestd, *gbesth, *gbestxd, *gbestxh, *lbesth, *lbestxh, *lbestd, *lbestxd;
	yh = (double*)malloc(sizeof(double) * 100000);
	gbesth = (double*)malloc(sizeof(double));
	gbestxh = (double*)malloc(sizeof(double)*N*P);
	lbestxh = (double*)malloc(sizeof(double)*SIZE);
	lbesth = (double*)malloc(sizeof(double)*POP);
	hipMalloc((void**)&yd, sizeof(double) * 100000);
	hipMalloc((void**)&gbestd, sizeof(double));
	hipMalloc((void**)&gbestxd, sizeof(double)*N*P);
	hipMalloc((void**)&lbestd, sizeof(double)*POP);
	hipMalloc((void**)&lbestxd, sizeof(double)*SIZE);

	double *rdh1, *rdh2, *rdh3;
	double *rdd1, *rdd2, *rdd3;
	rdh1 = (double*)malloc(sizeof(double) *SIZE);
	rdh2 = (double*)malloc(sizeof(double) *SIZE);
	rdh3 = (double*)malloc(sizeof(double) *SIZE);
	hipMalloc((void**)&rdd1, sizeof(double) * SIZE);
	hipMalloc((void**)&rdd2, sizeof(double) *SIZE);
	hipMalloc((void**)&rdd3, sizeof(double) * SIZE);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	srand((unsigned int)time(NULL));
	hiprandState* devStates;

	hipMalloc(&devStates, P*POP * sizeof(hiprandState));

	srand(time(0));
	int gen;
	int is_stoph, *is_stopd, stop_number = STOPNUM;
	hipMalloc((void**)&is_stopd, sizeof(int));
	for (gen = 0; gen < 1; gen++)
	{



		generate_s1(handle, sd1, initialxd, Id1);
		generate_s2(handle, sd2, initialxd, Id2);
		generate_t(handle, td, dd, initialxd);
		//�������ӹ��࣬�����Ż���
		parallel_updated_GPU << <BLOCKS, THREAD_NUM >> > (yd,objd, lbestd, lbestxd, initialxd, xd, dd, sd1, sd2, td, indexd, gen, devStates);//����ÿ��

		

	}

	hipMemcpy(gbestx, gbestxd, sizeof(double) * N*P, hipMemcpyDeviceToHost);
	hipMemcpy(y, yd, sizeof(double)*100000, hipMemcpyDeviceToHost);





	free(xh);
	free(indexh);
	free(sh1);
	free(sh2);
	free(Ih1);
	free(Ih2);
	free(th);
	free(dh);
	free(objh);
	free(initialxh);
	free(initialvh);
	free(yh);
	free(gbesth);
	free(gbestxh);
	free(lbesth);
	free(lbestxh);
	free(rdh1);
	free(rdh2);


	hipFree(xd);
	hipFree(indexd);
	hipFree(sd1);
	hipFree(sd2);
	hipFree(Id1);
	hipFree(Id2);
	hipFree(td);
	hipFree(dd);
	hipFree(objd);
	hipFree(initialxd);
	hipFree(initialvd);
	hipFree(yd);
	hipFree(gbestd);
	hipFree(gbestxd);
	hipFree(lbestd);
	hipFree(lbestxd);
	hipFree(rdd1);
	hipFree(rdd2);
	hipFree(is_stopd);
	hipblasDestroy(handle);
    return 0;
}